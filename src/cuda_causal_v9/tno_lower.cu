
#include <hip/hip_runtime.h>
#include<stdio.h>
#define B 2
#define N 16
#define D 32

template<typename F>
__global__ void lower_kernel(const int b, const int d, const int n, const F* T, const F* x, F* y) {
    /**
    input:
        T: d, n, [t0, t1, ..., t_(n-1)]
        x: b, d, n

    output:
        y: b, d, n
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int b_ = blockIdx.z * blockDim.z + threadIdx.z;
    if (b_ >= b) {
        return;
    }
    int t_offset = d_ * n;
    int x_offset = b_ * d * n + d_ * n;

    F s = 0;
    for (int j = 0; j <= i; j++) {
        s += T[t_offset + i - j] * x[x_offset + j];
    }
    y[x_offset + i] = s;
}

template<typename F>
__global__ void backward_kernel(const int b, const int d, const int n, const F* T, const F* x, const F* gy, F* gT, F* gx) {
    /**
    input:
        T: d, n, [t0, t1, ..., t_(n-1)]
        x: b, d, n
        gy: b, d, n

    output:
        gT: b, d, n
        gx: b, d, n
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int b_ = blockIdx.z * blockDim.z + threadIdx.z;
    if (b_ >= b) {
        return;
    }
    int t_offset = d_ * n;
    int x_offset = b_ * d * n + d_ * n;

    F s_x = 0;
    F s_T = 0;
    for (int j = 0; j < n - i; j++) {
        s_x += T[t_offset + j] * gy[x_offset + i + j];
        s_T += x[x_offset + j] * gy[x_offset + i + j];
    }
    gx[x_offset + i] = s_x;
    gT[x_offset + i] = s_T;
}

void forward_cuda(int b, int d, int n, float* T, float* x, float* y) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D, (b + B - 1) / B);
    dim3 DimBlock(N, D, B);
    lower_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, y);
}

void backward_cuda(int b, int d, int n, float* T, float* x, float* gy, float* gT, float* gx) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D, (b + B - 1) / B);
    dim3 DimBlock(N, D, B);
    backward_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, gy, gT, gx);
}