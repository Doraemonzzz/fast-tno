#include "hip/hip_runtime.h"
#include<stdio.h>
#define B 2
#define N 16
#define D 32

template<typename F>
__global__ void kernel(
    const int b, 
    const int d, 
    const int n, 
    const int s,
    const int e,
    const F* T, 
    const F* x, 
    F* y
) {
    /**
    input:
        T: d, n, [t_(-(n-1)), ..., t_(-1), t0, t_1, ..., t_(n-1)]
        x: b, d, n

    output:
        y: b, d, n
    **/
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j >= n) {
        return;
    }
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    if (i >= n) {
        return;
    }
    int idx = blockIdx.z * blockDim.z + threadIdx.z;
    if (idx >= b * d) {
        return;
    }
    // [-(n-1), (n-1)] -> [0, 2n-2]
    int i_block = blockIdx.y;
    int j_block = blockIdx.x;
    int i_thread = threadIdx.y;
    int j_thread = threadIdx.x;
    // number of block
    int l = (n + N - 1) / N;
    // block level index
    int k_block = i_block - j_block + l - 1;
    // thread level index
    int k_thread = i_thread - j_thread + N - 1;
    // batch global index
    int b_ = idx / d;
    int b_block = b_ / B;
    int b_thread = b_ % B;
    // feature global index
    int d_ = idx % d;
    int d_block = d_ / D;
    int d_thread = d_ % D;


    int start = max(s - i, 0);
    int end = min(e - i, n);
    // int i1 = blockIdx.x;
    // int j1 = blockIdx.y;
    int t_offset = d_ * n;
    int x_offset = b_ * d * n + d_ * n;

    // add bound check
    // __shared__ F T_shared[D][2 * N - 1];
    // __shared__ F x_shared[B][D][N];
    // T_shared[d_thread][k_thread] = T[t_offset + k_block * N + k_thread];
    // x_shared[b_thread][d_thread][k_thread] = x[x_offset + j];
    // __syncthreads();

    F tmp = 0;
    for (int u = 0; u < l; u++) {
        for (int v = 0; v < n; v++) {
            int j_ = u * N + v;
            if (j_ >= n) {
                break;
            }
            int k_block_ = i_block - u + l - 1;
            int k_thread_ = i_thread - v + N - 1;
            tmp += T[t_offset + k_block_ * N + k_thread_] * x[x_offset + j_];
        }
    }

    y[x_offset + i] = tmp;
}

template<typename F>
__global__ void backward_kernel(const int b, const int d, const int n, const F* T, const F* x, const F* gy, F* gT, F* gx) {
    /**
    input:
        T: d, n, [t0, t1, ..., t_(n-1)]
        x: b, d, n
        gy: b, d, n

    output:
        gT: b, d, n
        gx: b, d, n
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int b_ = blockIdx.z * blockDim.z + threadIdx.z;
    if (b_ >= b) {
        return;
    }
    int t_offset = d_ * n;
    int x_offset = b_ * d * n + d_ * n;

    F s_x = 0;
    F s_T = 0;
    for (int j = 0; j < n - i; j++) {
        s_x += T[t_offset + j] * gy[x_offset + i + j];
        s_T += x[x_offset + j] * gy[x_offset + i + j];
    }
    gx[x_offset + i] = s_x;
    gT[x_offset + i] = s_T;
}

void forward_cuda(int b, int d, int n, float* T, float* x, float* y) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D, (b + B - 1) / B);
    dim3 DimBlock(N, D, B);
    lower_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, y);
}

void backward_cuda(int b, int d, int n, float* T, float* x, float* gy, float* gT, float* gx) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D, (b + B - 1) / B);
    dim3 DimBlock(N, D, B);
    backward_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, gy, gT, gx);
}