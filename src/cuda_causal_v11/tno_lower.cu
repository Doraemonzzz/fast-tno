#include "hip/hip_runtime.h"
#include<stdio.h>
#define B 2
#define N 16
#define D 16
int l;

template<typename F>
__global__ void forward_kernel(
    const int b, 
    const int d, 
    const int n, 
    // const int l,
    const F* T, 
    const F* x, 
    F* y
) {
    /**
    input:
        T: d, n, [t_(-(n-1)), ..., t_(-1), t0, t_1, ..., t_(n-1)]
        x: b, d, n

    output:
        y: b, d, n
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int b_ = blockIdx.z * blockDim.z + threadIdx.z;
    if (b_ >= b) {
        return;
    }

    // int l = (n + N - 1) / N;
    int t_offset = d_ * n;
    int x_offset = b_ * d * n + d_ * n;

    int i_thread = threadIdx.x;
    int d_thread = threadIdx.y;
    int b_thread = threadIdx.z;

    __shared__ F T_shared[D][l][2 * N - 1];
    __shared__ F x_shared[B][D][N];

    for (int j_block = 0; j_block < l; j_block++) {
        for (int j_thread = 0; j_thread < N; j_thread++) {
            int j = j_block * N + j_thread;
            if (j >= n) {
                break;
            }
            int k = i - j;
            if (k >= n || k < 0) {
                break;
            }
            int k_thread = i_thread - j_thread + N - 1;
            T_shared[d_thread][j_block][k_thread] = T[t_offset + k];
            x_shared[b_thread][d_thread][j_thread] = x[x_offset + j];
        }
    }

    F s_y = 0;
    for (int j_block = 0; j_block < l; j_block++) {
        // j < n, 0 <= i - j < n
        // i - n < j <= min(i, n - 1)
        for (int j_thread = 0; j_thread < N; j_thread++) {
            int j = j_block * N + j_thread;
            if (j >= n) {
                break;
            }
            int k = i - j;
            if (k >= n || k < 0) {
                break;
            }
            int k_thread = i_thread - j_thread + N - 1;
            s_y += T_shared[d_thread][j_block][k_thread] * x_shared[b_thread][d_thread][j_thread];
        }
    }

    y[x_offset + i] = s_y;
}

template<typename F>
__global__ void backward_kernel(
    const int b, 
    const int d, 
    const int n, 
    // const int s,
    // const int e,
    const F* T, 
    const F* x, 
    const F* gy, 
    F* gT, 
    F* gx
) {
    /**
    input:
        T: d, n, [t0, t_1, ..., t_(n-1)]
        x: b, d, n, [x0, x_1, ..., x_(n-1)]
        gy: b, d, n

    output:
        gT: b, d, n
        gx: b, d, n
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int b_ = blockIdx.z * blockDim.z + threadIdx.z;
    if (b_ >= b) {
        return;
    }
    // number of block
    int l = (n + N - 1) / N;
    int t_offset = d_ * n;
    int x_offset = b_ * d * n + d_ * n;

    F s_x = 0;
    F s_T = 0;
    for (int j_block = 0; j_block < l; j_block++) {
        for (int j_thread = 0; j_thread < N; j_thread++) {
            int j = j_block * N + j_thread;
            if (j >= n) {
                break;
            }
            int k = j - i;
            if (k >= n || k < 0) {
                continue;
            }

            s_x += T[t_offset + k] * gy[x_offset + j];
            s_T += x[x_offset + k] * gy[x_offset + j];
        }
    }

    gx[x_offset + i] = s_x;
    gT[x_offset + i] = s_T;
}

void forward_cuda(int b, int d, int n, float* T, float* x, float* y) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D, (b + B - 1) / B);
    dim3 DimBlock(N, D, B);
    int l = (n + N - 1) / N;
    forward_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, y);
}

void backward_cuda(int b, int d, int n, float* T, float* x, float* gy, float* gT, float* gx) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D, (b + B - 1) / B);
    dim3 DimBlock(N, D, B);
    backward_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, gy, gT, gx);
}