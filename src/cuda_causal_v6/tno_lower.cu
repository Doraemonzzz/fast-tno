
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 32
#define D 32

template<typename F>
__global__ void lower_kernel(const int b, const int d, const int n, const F* T, const F* x, F* y) {
    /**
    input:
        T: n, d [t0, t1, ..., t_(n-1)]
        x: b, n, d

    output:
        y: b, n, d
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int t_offset = d_;

    for (int b_ = 0; b_ < b; b_++) {
        F s = 0;
        int x_offset = b_ * d * n + d_;
        for (int j = 0; j <= i; j++) {
            s += T[t_offset + (i - j) * d] * x[x_offset + j * d];
        }
        y[x_offset + i * d] = s;
    }
}

template<typename F>
__global__ void backward_kernel(const int b, const int d, const int n, const F* T, const F* x, const F* gy, F* gT, F* gx) {
    /**
    input:
        T: n, d, [t0, t1, ..., t_(n-1)]
        x: b, n, d
        gy: b, n, d

    output:
        gT: b, n, d
        gx: b, n, d
    **/
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    int d_ = blockIdx.y * blockDim.y + threadIdx.y;
    if (d_ >= d) {
        return;
    }
    int t_offset = d_;

    for (int b_ = 0; b_ < b; b_++) {
        int x_offset = b_ * d * n + d_;
        F s_x = 0;
        F s_T = 0;
        for (int j = 0; j < n - i; j++) {
            s_x += T[t_offset + j * d] * gy[x_offset + (i + j) * d];
            s_T += x[x_offset + j * d] * gy[x_offset + (i + j) * d];
        }
        gx[x_offset + i * d] = s_x;
        gT[x_offset + i * d] = s_T;
    }
}

void forward_cuda(int b, int d, int n, float* T, float* x, float* y) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D);
    dim3 DimBlock(N, D);
    lower_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, y);
}

void backward_cuda(int b, int d, int n, float* T, float* x, float* gy, float* gT, float* gx) {
    dim3 DimGrid((n + N - 1) / N, (d + D - 1) / D);
    dim3 DimBlock(N, D);
    backward_kernel<<<DimGrid, DimBlock>>>(b, d, n, T, x, gy, gT, gx);
}